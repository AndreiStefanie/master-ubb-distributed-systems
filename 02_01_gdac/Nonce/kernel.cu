#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>

#include "config.h"
#include "fmt.h"

// SHA1 Implementation
typedef struct {
	BYTE data[64];
	WORD datalen;
	LONG bitlen;
	WORD state[5];
	WORD k[4];
} CUDA_SHA1_CTX;

#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif

__device__  __forceinline__ void cuda_sha1_transform(CUDA_SHA1_CTX* ctx, const BYTE data[])
{
	WORD a, b, c, d, e, i, j, t, m[80];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + (data[j + 3]);
	for (; i < 80; ++i) {
		m[i] = (m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16]);
		m[i] = (m[i] << 1) | (m[i] >> 31);
	}

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	for (i = 0; i < 20; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + ctx->k[0] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 40; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[1] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 60; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d)) + e + ctx->k[2] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for (; i < 80; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[3] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}

__device__ void cuda_sha1_init(CUDA_SHA1_CTX* ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;
	ctx->k[0] = 0x5a827999;
	ctx->k[1] = 0x6ed9eba1;
	ctx->k[2] = 0x8f1bbcdc;
	ctx->k[3] = 0xca62c1d6;
}

__device__ void cuda_sha1_update(CUDA_SHA1_CTX* ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			cuda_sha1_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void cuda_sha1_final(CUDA_SHA1_CTX* ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_sha1_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	cuda_sha1_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and MD uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

__device__ BYTE* d_strcpy(BYTE* dest, BYTE* src) {
	int i = 0;

	do {
		dest[i] = src[i];
	} while (src[i++] != 0);

	return dest;
}

__device__ void d_reverse(BYTE str[], int size)
{
	int start = 0;
	int end = size - 1;
	while (start < end) {
		char h = *(str + start), t = *(str + end);
		*(str + start) = t;
		*(str + end) = h;
		start++;
		end--;
	}
}

__device__ BYTE* d_itob(size_t num, BYTE* str, int* size)
{
	int i = 0;

	/* Handle 0 explicitely, otherwise empty string is printed for 0 */
	if (num == 0) {
		str[i++] = '0';
		return str;
	}

	// Process individual digits 
	while (num != 0) {
		int rem = num % 10;
		str[i++] = (rem > 9) ? (rem - 10) + 'a' : rem + '0';
		num = num / 10;
	}

	// Reverse the string 
	d_reverse(str, i);
	*size = i;

	return str;
}

__device__ BYTE* d_strcat(BYTE* dest, BYTE* src) {
	int i = 0;

	while (dest[i] != 0) {
		i++;
	}
	d_strcpy(dest + i, src);

	return dest;
}

__device__ void makedigits(BYTE x, BYTE(&digits)[2])
{
	BYTE d0 = x / 16;
	digits[1] = x - d0 * 16;
	BYTE d1 = d0 / 16;
	digits[0] = d0 - d1 * 16;
}

__device__ void makehex(BYTE(&digits)[2], char(&hex)[2])
{
	for (int i = 0; i < 2; ++i) {
		if (digits[i] < 10) {
			hex[i] = '0' + digits[i];
		}
		else {
			hex[i] = 'a' + (digits[i] - 10);
		}
	}
}

// The kernel
__global__ void find_nonce(size_t* result, BYTE* hash, bool* found, size_t stride) {
	CUDA_SHA1_CTX ctx;
	BYTE checksum[SHA_SIZE];

	unsigned int thread = blockIdx.x * blockDim.x + threadIdx.x;
	size_t nonce_source = thread + stride;

	// Prepare the input
	int nonce_size = 0;
	BYTE nonce[SHA_SIZE];
	BYTE input[BUFFER_SIZE + SHA_SIZE];

	d_itob(nonce_source, nonce, &nonce_size);
	memcpy(input, (BYTE*)BUFFER, BUFFER_SIZE);
	memcpy(input + BUFFER_SIZE, nonce, nonce_size);

	memset(checksum, 0x0, SHA_SIZE);

	cuda_sha1_init(&ctx);
	cuda_sha1_update(&ctx, input, BUFFER_SIZE + nonce_size);
	cuda_sha1_final(&ctx, checksum);

	bool suffix_matches = true;
	for (int i = 0; i < ZEROS_TO_FIND; i++) {
		if (checksum[SHA_SIZE - i - 1] != 0) {
			suffix_matches = false;
			break;
		}
	}

	if (suffix_matches) {
		*found = true;
		*result = nonce_source;
		d_strcpy(hash, checksum);
	}
}

void get_gpu_props(int* prop)
{
	hipDeviceProp_t deviceProp;

	prop[0] = 0;
	prop[1] = 0;

	if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0))
	{
		prop[0] = 32;
		prop[1] = 32;
		return;
	}
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&prop[0], find_nonce, deviceProp.warpSize, 0);
	prop[1] = deviceProp.warpSize;
}

int main(int argc, char** argv) {
	bool h_found = false;
	size_t h_nonce = 0;
	size_t nonce_size = sizeof(size_t);
	int gpu_props[2];

	size_t i = 0;
	size_t stride = 0;
	size_t th_count = 0;

	struct timeb start, end;
	double seconds = 0;

	hipError_t status = hipSuccess;

	get_gpu_props(gpu_props);

	// Initialize the input data
	BYTE* h_digest = (BYTE*)malloc(SHA_SIZE);
	memset(h_digest, 0, SHA_SIZE);

	// Initialize the device variables
	size_t* d_nonce;
	bool* d_found;
	BYTE* d_digest;
	status = hipMalloc((void**)&d_nonce, nonce_size);
	if (hipSuccess != status) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	status = hipMalloc((void**)&d_digest, SHA_SIZE);
	if (hipSuccess != status) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	status = hipMalloc((void**)&d_found, sizeof(bool));
	if (hipSuccess != status) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	status = hipMemcpy(d_found, &h_found, sizeof(bool), hipMemcpyHostToDevice);
	if (hipSuccess != status) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Start the timer
	ftime(&start);

	th_count = gpu_props[0] * gpu_props[1];
	do {
		find_nonce << <gpu_props[0], gpu_props[1] >> > (d_nonce, d_digest, d_found, stride);
		status = hipGetLastError();
		if (hipSuccess != status) {
			fprintf(stderr, "Failed to launch the kernel!");
			goto Error;
		}

		status = hipDeviceSynchronize();
		if (hipSuccess != status) {
			fprintf(stderr, "hipDeviceSynchronize failed!");
			goto Error;
		}

		status = hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
		if (hipSuccess != status) {
			fprintf(stderr, "Failed to copy the found bool back to host");
			goto Error;
		}

		stride += th_count;
		i++;
	} while (!h_found && i <= MAX_ITERATIONS);


	// Copy the data back to the host
	status = hipMemcpy(h_digest, d_digest, SHA_SIZE, hipMemcpyDeviceToHost);
	if (hipSuccess != status) {
		fprintf(stderr, "Failed to copy the resulting hash back to host");
		goto Error;
	}
	status = hipMemcpy(&h_nonce, d_nonce, nonce_size, hipMemcpyDeviceToHost);
	if (hipSuccess != status) {
		fprintf(stderr, "Failed to copy the found nonce back to host");
		goto Error;
	}

	// Stop the timer
	ftime(&end);
	seconds = end.time - start.time + ((double)end.millitm - (double)start.millitm) / 1000.0;

	printf("Hashrate: %s hashes/s. Duration: %.2f seconds\n", fmt_num((size_t)(stride / seconds)).c_str(), seconds);

	if (true == h_found) {
		char hex_result[SHA_SIZE * 2 + 1]{};
		for (int offset = 0; offset < SHA_SIZE; offset++) {
			sprintf((hex_result + (2 * offset)), "%02x", h_digest[offset] & 0xff);
		}
		printf("Nonce: %lld. Digest: %s\n", h_nonce, hex_result);
	}
	else {
		printf("Could not find nonce such that the digest ends in %d zeros\n", ZEROS_TO_FIND);
	}

Error:
	free(h_digest);
	hipFree(d_nonce);
	hipFree(d_digest);

	return status;
}