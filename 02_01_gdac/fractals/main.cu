#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "util.h"

using namespace std;

__constant__ unsigned char d_colormap[(MAX_ITERATION + 1) * 3];

__global__ void generate_image(unsigned char* image, int width, int height, int max) {
	unsigned int x_dim = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_dim = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int index = 4 * width * y_dim + 4 * x_dim;

	double c_re = (x_dim - width / 2.0) * 4.0 / width;
	double c_im = (y_dim - height / 2.0) * 4.0 / width;

	double x = 0.0;
	double y = 0.0;

	unsigned int iteration = 0;

	while (x * x + y * y <= 4 && iteration < max) {
		double x_new = x * x - y * y + c_re;
		y = 2 * x * y + c_im;
		x = x_new;
		iteration++;
	}

	if (iteration > max) {
		iteration = max;
	}

	unsigned char* c = &d_colormap[iteration * 3];
	image[index + 0] = c[0];
	image[index + 1] = c[1];
	image[index + 2] = c[2];
	image[index + 3] = 255;
}

int main(int argc, char** argv) {
	struct timeb start, end;
	double times[REPEAT];
	char path[255];
	hipError_t status = hipSuccess;

	size_t colormap_size = (MAX_ITERATION + 1) * 3;
	size_t image_size = WIDTH * HEIGHT * 4;

	dim3 blockDim(THREADS_X, THREADS_Y, 1);
	dim3 gridDim(WIDTH / blockDim.x, HEIGHT / blockDim.y, 1);

	// Initialize the host image and colormap
	unsigned char* h_colormap = (unsigned char*)malloc(colormap_size);
	unsigned char* h_image = (unsigned char*)malloc(image_size);

	init_colormap(MAX_ITERATION, h_colormap);

	unsigned char* d_image;
	status = hipMalloc((void**)&d_image, image_size);
	if (hipSuccess != status) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	for (int i = 0; i < REPEAT; i++)
	{
		memset(h_image, 0, image_size);

		// Start the timer, including the copy of the image between the host and the device
		ftime(&start);

		status = hipMemcpyToSymbol(HIP_SYMBOL(d_colormap), h_colormap, colormap_size);
		if (hipSuccess != status) {
			fprintf(stderr, "Failed to copy colormap to device");
			goto Error;
		}

		generate_image << <gridDim, blockDim >> > (d_image, WIDTH, HEIGHT, MAX_ITERATION);

		status = hipGetLastError();
		if (hipSuccess != status) {
			fprintf(stderr, "generate_image failed!");
			goto Error;
		}

		status = hipDeviceSynchronize();
		if (hipSuccess != status) {
			fprintf(stderr, "hipDeviceSynchronize failed!");
			goto Error;
		}

		status = hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
		if (hipSuccess != status) {
			fprintf(stderr, "Failed to copy image back to host");
			goto Error;
		}

		ftime(&end);
		times[i] = end.time - start.time + ((double)end.millitm - (double)start.millitm) / 1000.0;

		sprintf(path, IMAGE, "gpu", i);
		save_image(path, h_image, WIDTH, HEIGHT);
		progress("gpu", i, times[i]);
	}

	report("gpu", times);

Error:
	free(h_image);
	free(h_colormap);
	hipFree(d_image);

	return status;
}